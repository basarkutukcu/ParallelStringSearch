#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <windows.h>
#include <iostream>
#include <fstream>
#include <sstream>

double PCFreq = 0.0;
__int64 CounterStart = 0;

void StartCounter()
{
	LARGE_INTEGER li;
	if (!QueryPerformanceFrequency(&li))
		std::cout << "QueryPerformanceFrequency failed!\n";

	PCFreq = double(li.QuadPart) / 1000.0;

	QueryPerformanceCounter(&li);
	CounterStart = li.QuadPart;
}
double GetCounter()
{
	LARGE_INTEGER li;
	QueryPerformanceCounter(&li);
	return double(li.QuadPart - CounterStart) / PCFreq;
}

__global__ void parallelGrep(char *global_data, int globalData_Size, char *key, int key_size, int *key_indexes, int *curr_index)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	int key0 = key[0];

	if (idx < globalData_Size)
	{
		if (global_data[idx] == key0)
		{
			int save = 1;

			for (int i = 1; i < key_size; i++)
			{
				if (global_data[idx + i] != key[i])
				{
					save = 0;
					break;
				}
			}
			if (save == 1)
			{
				key_indexes[atomicAdd(curr_index, 1)] = idx;
			}
		}
	}

}

int main()
{
	FILE *pFile;
	size_t numB;
	long lSize;
	size_t key_size, key_ind_size;
	char *data_d, *data_h, *key_d;
	char key_h[50];
	int key_ind_h[600];
	int *key_ind_d;
	int currind;
	int *curr_ind_h = &currind;
	int *curr_ind_d;



	/*******************************Data and Data Size*********************************************************/
	// When searching Bilbo in OriginalText, set key_ind_h[600].
	// When searching 'Bilbo' in EditedText, set key_ind_h[12000].
	pFile = fopen("OriginalText.txt", "rb");
	if (pFile == NULL)
	{
		printf("Cannot open txt file!\n");
		exit(1);
	}
	fseek(pFile, 0, SEEK_END);
	lSize = ftell(pFile);
	rewind(pFile);

	data_h = new char[lSize];	// allocate memory on host
	hipMalloc((void **)&data_d, lSize);	// allocate memory on device	- this is the first CUDA call so it takes huge amount of time

	StartCounter();

	numB = fread(data_h, 1, lSize, pFile);
	hipMemcpy(data_d, data_h, lSize, hipMemcpyHostToDevice);	// copy data to device memory
	/************************************************************************************************************/

	/******************************key and key_size*************************************************************/
	//printf("What do you want to search: ");
	//scanf("%[^\n]s ", key_h);	// enter the key
	strcpy(key_h, "Bilbo");
	key_size = strlen(key_h);

	//StartCounter();

	hipMalloc((void **)&key_d, key_size);
	hipMemcpy(key_d, key_h, key_size, hipMemcpyHostToDevice);
	/**********************************************************************************************************/

	/****************************key indices and current indice*************************************************/
	key_ind_size = sizeof(key_ind_h);
	memset(key_ind_h, 0, key_ind_size);
	hipMalloc((void **)&key_ind_d, key_ind_size);
	hipMemcpy(key_ind_d, key_ind_h, key_ind_size, hipMemcpyHostToDevice);

	*curr_ind_h = 0;
	hipMalloc((void **)&curr_ind_d, 4);
	hipMemcpy(curr_ind_d, curr_ind_h, 4, hipMemcpyHostToDevice);
	/***********************************************************************************************************/

	int block_size = 1024;
	int n_blocks = lSize / block_size + (lSize%block_size == 0 ? 0 : 1);

	parallelGrep << < n_blocks, block_size>> > (data_d, lSize, key_d, key_size, key_ind_d, curr_ind_d);
	
	// For debug
	/*hipError_t err;
	err = hipPeekAtLastError();*/

	hipMemcpy(key_ind_h, key_ind_d, key_ind_size, hipMemcpyDeviceToHost);


	printf("\n Elapsed Time: ");
	std::cout << GetCounter() << " ms" << std::endl;


	/*****************	To print lines with key string	************************/
	//int firstChar;
	//int pIter = 0;
	//while (key_ind_h[pIter] != 0)
	//{
	//	firstChar = key_ind_h[pIter];
	//	// Find the first character of the line
	//	while (data_h[firstChar - 1] != '\n')
	//	{
	//		firstChar--;
	//	}
	//	// print till the last character of the line
	//	while (data_h[firstChar] != '\n')
	//	{
	//		printf("%c", data_h[firstChar]);
	//		firstChar++;
	//	}
	//	printf("\n");
	//	pIter++;
	//}
	//printf("\n Elapsed Time: ");
	//std::cout << GetCounter() <<" ms"<< std::endl;
	/*****************************************************************************/

	delete[] data_h;
	hipFree(data_d);
	hipFree(key_d);
	hipFree(key_ind_d);
	hipFree(curr_ind_d);
	return 0;
}
